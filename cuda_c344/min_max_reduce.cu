#include "hip/hip_runtime.h"
#include <cmath>
#include <stdio.h>
#include <stdlib.h>
#include "utils.h"
#include <time.h>

// cuda-memcheck ./a.out debug segfault
// nvcc -arch=sm_21 min_max_reduce.cu -G0 # compile
#define H_MIN(a,b) (((a)<(b))?(a):(b))
#define H_MAX(a,b) (((a)>(b))?(a):(b))


typedef float (*reduce_cb) (float &, float &);

__device__ float MIN(float &x, float &y)
{
    return x < y ? x : y;
}

__device__ float MAX(float &x, float &y)
{
    return x > y ? x : y;
}

template<reduce_cb cb>
__global__ void reduce(
    float *input, float *output, int *n, int *nRows, int *nCols, int *blocksY)
{
    /**
    __shared__ temp has a max size of 49152 b
    so the blokcs are split accordingly
    So now we need x values writen in outpu where x=blockSize.x * blockSize.y
    then we need to launch another kernel with input arr[x]
    Every block executes 1 blelloch
    **/
    extern __shared__ float temp[];// allocated on invocation
    __shared__ float last_elm;

    const int2 thread_2D_pos = make_int2( blockIdx.x * blockDim.x + threadIdx.x,
                                          blockIdx.y * blockDim.y + threadIdx.y);

    const int thid = thread_2D_pos.y * ( *nCols ) + thread_2D_pos.x;
    const int b_thid = threadIdx.x;
    int offset = 1;

    if (2 * thid + 1 >= *n)
    {
        return;
    }
    temp[b_thid] = input[2 * thid]; // load input into shared memory
    temp[b_thid + 1] = input[2 * thid + 1];

    __syncthreads();
    // build sum in place up the tree
    for (int d = 1024 >> 1; d > 0; d >>= 1)
    {
        __syncthreads();
        if (b_thid < d)
        {
            const int ai = offset * (b_thid + 1) - 1;
            const int bi = offset * (b_thid + 2) - 1;
            temp[bi] = cb(temp[ai], temp[bi]);
        }
        offset *= 2;
    }
    // clear the last element
    if (b_thid == 0)
    {
        last_elm = temp[1023];
        temp[1023] = 0.f;
    }
    //offset = *n;
    // traverse down tree & build scan
    __syncthreads();
    for (int d = 1; d < 1024; d *= 2)
    {
        offset >>= 1;
        __syncthreads();
        if (b_thid < d)
        {
            int ai = offset * (b_thid + 1) - 1;
            int bi = offset * (b_thid + 2) - 1;

            if (ai >= 0 && bi >= 0)
            {
                float swap_temp = temp[ai];
                temp[ai] = temp[bi];
                temp[bi] = cb(swap_temp, temp[bi]);
            }
        }
    }
    __syncthreads();
    // write results to device memory
    if (b_thid == 0)
    {
        int index = (blockIdx.x * (*blocksY ) + blockIdx.y);
        output[index] = cb(temp[1023], last_elm);
    }
}


int shared_memory_per_block()
{
    int nDevices;
    hipGetDeviceCount(&nDevices);
    int sharedMemPerBlock = -1;

    for (int i = 0; i < nDevices; i++)
    {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        sharedMemPerBlock = prop.sharedMemPerBlock;
    }
    return sharedMemPerBlock;
}

int main(int argc, char **argv)
{
    int numCols = 1536;
    int numRows = 1024;

    int ARRAY_SIZE = numCols * numRows;
    int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

    time_t rand_t;
    srand((unsigned) time(&rand_t));

    float *h_in =  (float *)malloc(ARRAY_BYTES);
    float *h_out = (float *)malloc(ARRAY_BYTES + sizeof(float));

    for (int i = 0; i < ARRAY_SIZE; i++)
    {
        h_in[i] = rand() % 20000;
    }


    float *d_in;
    float *d_out;
    int _h_in = ARRAY_SIZE;
    int *h_n = &_h_in;
    int *d_n, *nRows, *nCols, *blocksY;

    // this is the amount of __shared__ we can use
    int sharedMemPerBlock = shared_memory_per_block();
    // Split kernels to match the cache size in a square
    int blocks = (int)floor(sqrt(ARRAY_BYTES / sharedMemPerBlock));

    checkCudaErrors(hipMalloc((void **) &d_in, ARRAY_BYTES));
    checkCudaErrors(hipMalloc((void **) &d_out, ARRAY_BYTES + sizeof(float)));
    checkCudaErrors(hipMalloc((void **) &d_n, sizeof(int)));
    checkCudaErrors(hipMalloc((void **) &nCols, sizeof(int)));
    checkCudaErrors(hipMalloc((void **) &nRows, sizeof(int)));
    checkCudaErrors(hipMalloc((void **) &blocksY, sizeof(int)));

    checkCudaErrors(hipMemcpy(
        d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(
        d_n, h_n, sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(
        nRows, &numRows, sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(
        nCols, &numCols, sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(
        blocksY, &blocks, sizeof(int), hipMemcpyHostToDevice));

    dim3 blockSize(blocks, blocks, 1);
    // good luck here +++
    reduce<MAX> <<< blockSize, 1024, 1024 * sizeof(float)>>>(
        d_in, d_out, d_n, nRows, nCols, blocksY);
    hipDeviceSynchronize();

    // check for error
    hipError_t error = hipGetLastError();
    if (error != hipSuccess)
    {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    checkCudaErrors(hipMemcpy(
        h_out, d_out, ARRAY_BYTES + sizeof(float), hipMemcpyDeviceToHost));

    // hipMemcpyFromSymbol(&h_output, HIP_SYMBOL("d_output"), sizeof(float), 0, hipMemcpyDeviceToHost);

    float max = -1;
    for (int i = 0; i < ARRAY_SIZE; i++)
    {
        if (h_in[i] > max)
        {
            max = h_in[i];
        }
    }
    printf("%f max\n", max);
    printf("%f last elm\n", h_out[ARRAY_SIZE]);

    checkCudaErrors(hipFree(d_in));
    checkCudaErrors(hipFree(d_n));
    checkCudaErrors(hipFree(d_out));

    return 0;
}
//http://blog.codinghorror.com/content/images/uploads/2008/08/6a0120a85dcdae970b012877705d12970c-pi.jpg
