int nDevices;
hipGetDeviceCount(&nDevices);
int = sharedMemPerBlock;

for (int i = 0; i < nDevices; i++)
{
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    sharedMemPerBlock = prop.sharedMemPerBlock;
}
